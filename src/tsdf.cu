#include "tsdf_base.h"

int main(){
	//string file_list = "..//data//boxes_NYU_po_test_nb2000_fb.list";
	string file_list = "..//data//boxes_SUNrgbd_po_test_nb2000_fb.list";
    	string data_root =  "..//data//";
	string output_data = "..//data//julia_data//";
	int maxscenes = 5;

	std::vector<Scene3D*> scenes;
	std::vector<int> box_id;

	int totalObjectCount = 0;
	float scale =100;
	float context_pad =3;
	std::vector<int> grid_size {3,30,30,30};
	int encode_type =100;

	int totalScenes = 0;

	std::cout<<"loading file "<<file_list<<"\n";
	FILE* fp = fopen(file_list.c_str(),"rb");
	if (fp==NULL) { std::cout<<"fail to open file: "<<file_list<<std::endl; exit(EXIT_FAILURE); }
	while (feof(fp)==0 && totalScenes < maxscenes) 
	{

		Scene3D* scene = new Scene3D();
		unsigned int len = 0;
		fread((void*)(&len), sizeof(unsigned int), 1, fp);    
		if (len==0) break;
		scene->filename.resize(len);
		if (len>0) fread((void*)(scene->filename.data()), sizeof(char), len, fp);

		//TSDF file
		int lastback = scene->filename.find_last_of("/");
		string outputname = "";
		if (lastback > 0)
		{
			outputname = scene->filename.substr(lastback+1);
		}
		else
		{
			FatalError(__LINE__);
			continue;
		}
		scene->filename = data_root+scene->filename+".bin"; 
		string tsdffile = output_data+outputname+".tsdf";


		fread((void*)(scene->R), sizeof(float), 9, fp);
		fread((void*)(scene->K), sizeof(float), 9, fp);
		fread((void*)(&scene->height), sizeof(unsigned int), 1, fp);  
		fread((void*)(&scene->width), sizeof(unsigned int), 1, fp); 


		fread((void*)(&len),    sizeof(unsigned int),   1, fp);
		scene->objects.resize(len);
		if (len>0){
		  totalObjectCount += len;
		  for (int i=0;i<len;++i){
		      Box3D box;
		      fread((void*)(&(box.category)), sizeof(unsigned int),   1, fp);
		      fread((void*)(box.base),        sizeof(float), 9, fp);
		      fread((void*)(box.center),      sizeof(float), 3, fp);
		      fread((void*)(box.coeff),       sizeof(float), 3, fp);
		      //process box pad contex oreintation 
		      box = processbox (box, context_pad, grid_size[1]);
		      scene->objects[i]=box;
		      box_id.push_back(i);
		  }
		}
		scenes.push_back(scene);
		totalScenes++;

		float* dataCPUmem = new float[(len)*3*30*30*30];
		StorageT* dataGPUmem;
		checkCUDA(__LINE__, hipMalloc(&dataGPUmem, (len)*3*30*30*30*sizeof(float)));
		compute_TSDF(&scenes, &box_id, dataGPUmem,grid_size,encode_type,scale);
		checkCUDA(__LINE__, hipMemcpy(dataCPUmem, dataGPUmem,(len)*3*30*30*30*sizeof(float), hipMemcpyDeviceToHost) );

		//clear for workaround
		scenes.clear();
		box_id.clear();

		//can print these or not
		//totalScenes
		//totalObjectCount

		FILE * fid = fopen(tsdffile.c_str(),"wb");
		fwrite(dataCPUmem,sizeof(float),(len)*3*30*30*30,fid);
		fclose(fid);

		cout << "Scene: " << totalScenes << " Boxes: " << len << " Bin: " << scene->filename << " TSDF: " << tsdffile << endl;
		
		//free
		delete scene;
		delete[] dataCPUmem;
		//delete dataGPUmem;
		hipFree(dataGPUmem);
	}
	fclose(fp);
	return 1;
}
