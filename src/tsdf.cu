//Include
#include "tsdf.h"

//Global variables
std::vector<Scene3D*> scenes;
std::vector<int> box_id;
int totalObjectCount = 0;
float scale = 100;
float context_pad =3;
std::vector<int> grid_size {3,30,30,30};
int encode_type =100;
int totalScenes = 0;
string file_list = "";
string data_root =  "..//data//";
string output_data = "..//data//julia_data//";
int maxscenes = 0;
//list file
FILE* fp = NULL;

//Export functions to be called from Julia
extern "C"
{
	//Initialize variables
	void initTSDF(int db, int maxs)
	{
		if (db == 0)
			file_list = "..//data//boxes_NYU_po_test_nb2000_fb.list";
		else
			file_list = "..//data//boxes_SUNrgbd_po_test_nb2000_fb.list";

		cout << "Loading file: " << file_list << endl << endl;
		fp = fopen(file_list.c_str(),"rb");
		if (fp==NULL) { cout << "Failed to open file: "<< file_list << endl; exit(EXIT_FAILURE); }
		maxscenes = maxs;
	}

	//Call this when done with TSDF
	void freeTSDF()
	{
		fclose(fp);
	}

	//Compute TSDF for the next scene
	int getNextTSDF(int& count)
	{
		if (feof(fp)==0 && totalScenes < maxscenes)
		{
			Scene3D* scene = new Scene3D();
			unsigned int len = 0;
			fread((void*)(&len), sizeof(unsigned int), 1, fp);    
			if (len==0) return -1;
			scene->filename.resize(len);
			if (len>0) fread((void*)(scene->filename.data()), sizeof(char), len, fp);

			//TSDF file
			string tsdffile = output_data+"temp.tdsf";
			scene->filename = data_root+scene->filename+".bin";

			fread((void*)(scene->R), sizeof(float), 9, fp);
			fread((void*)(scene->K), sizeof(float), 9, fp);
			fread((void*)(&scene->height), sizeof(unsigned int), 1, fp);  
			fread((void*)(&scene->width), sizeof(unsigned int), 1, fp); 


			fread((void*)(&len),    sizeof(unsigned int),   1, fp);
			scene->objects.resize(len);
			if (len>0){
			  totalObjectCount += len;
			  for (int i=0;i<len;++i){
			      Box3D box;
			      fread((void*)(&(box.category)), sizeof(unsigned int),   1, fp);
			      fread((void*)(box.base),        sizeof(float), 9, fp);
			      fread((void*)(box.center),      sizeof(float), 3, fp);
			      fread((void*)(box.coeff),       sizeof(float), 3, fp);
			      //process box pad contex oreintation 
			      box = processbox (box, context_pad, grid_size[1]);
			      scene->objects[i]=box;
			      box_id.push_back(i);
			  }
			}
			scenes.push_back(scene);
			totalScenes++;

			cout << "Scene: " << totalScenes << " Boxes: " << len << " Bin: " << scene->filename << " TSDF: " << tsdffile << endl << endl;

			unsigned long long  time0,time1;

			time0 = get_timestamp_dss();
			float* dataCPUmem = new float[len*3*30*30*30];
			StorageT* dataGPUmem;
			checkCUDA(__LINE__, hipMalloc(&dataGPUmem, (len)*3*30*30*30*sizeof(float)));
			time1 = get_timestamp_dss();
			cout << "cpu->gpu time " << (time1-time0)/1000 << " ms" << endl;

			time0 = get_timestamp_dss();
			compute_TSDF(&scenes, &box_id, dataGPUmem,grid_size,encode_type,scale);
			time1 = get_timestamp_dss();
			cout << "compute time " << (time1-time0)/1000 << " ms" << endl;

			time0 = get_timestamp_dss();
			checkCUDA(__LINE__, hipMemcpy(dataCPUmem, dataGPUmem,(len)*3*30*30*30*sizeof(float), hipMemcpyDeviceToHost) );
			time1 = get_timestamp_dss();
			cout << "gpu->cpu time " << (time1-time0)/1000 << " ms" << endl;

			//clear for workaround
			scenes.clear();
			box_id.clear();

			//write TSDF to temp file because couldnt figure out how to pass it to Julia
			time0 = get_timestamp_dss();
			FILE * fid = fopen(tsdffile.c_str(),"wb");
			fwrite(dataCPUmem,sizeof(float),len*3*30*30*30,fid);
			fclose(fid);
			time1 = get_timestamp_dss();
			cout << "cpu->file " << (time1-time0)/1000 << " ms" << endl << endl;

			//Pass by reference paramater update
			count = len;
	
			//free memory
			delete scene;
			delete[] dataCPUmem;
			hipFree(dataGPUmem);
			return 1;
		}
		return -1;
	}
}




void convertBoxesList()
{
	string box2d = data_root+"boxes2d_NYU_po_nb2000.list";
    	cout<<"Loading 2d boxes list file: "<< box2d << endl;
    	FILE* fp2d = fopen(box2d.c_str(),"rb");
    	if (fp2d==NULL) { cout << "Failed to open file: "<< box2d<< endl; exit(EXIT_FAILURE); }


    	while (feof(fp2d)==0) {
      		Scene3D* scene = new Scene3D();
      		unsigned int len = 0;
      		size_t file_size = 0;
      		file_size += fread((void*)(&len), sizeof(unsigned int), 1, fp2d);    
      		if (len==0) break;
      		scene->filename.resize(len);
      		if (len>0) file_size += fread((void*)(scene->filename.data()), sizeof(char), len, fp2d);
     
		int inx = scene->filename.find_last_of("/");
		string output="Boxes//"+scene->filename.substr(inx+1)+".txt";
		FILE* myfile = fopen(output.c_str(), "w");
	

	      	file_size += fread((void*)(scene->R), sizeof(float), 9, fp2d);
		file_size += fread((void*)(scene->K), sizeof(float), 9, fp2d);
		file_size += fread((void*)(&scene->height), sizeof(unsigned int), 1, fp2d);
		file_size += fread((void*)(&scene->width), sizeof(unsigned int), 1, fp2d); 
		file_size += fread((void*)(&len),    sizeof(unsigned int),   1, fp2d);
		scene->objects.resize(len);
      
      		//cout<<scene->filename << endl;
      		for (int bid = 0;bid<len;bid++){
			/*
			struct Box2D{
			  unsigned int category;
			  float tblr[4];
			};
			*/
			Box2D box;
			file_size += fread((void*)(&(box.category)), sizeof(unsigned int),   1, fp2d);
			file_size += fread((void*)(box.tblr),        sizeof(float), 4, fp2d);
			scene->objects_2d_tight.push_back(box);
			 

			fprintf(myfile, "%d %f %f %f %f\n", box.category, box.tblr[0], box.tblr[1], box.tblr[2], box.tblr[3]);


		 	uint8_t hasTarget = 0;
			file_size += fread((void*)(&hasTarget), sizeof(uint8_t),   1, fp2d);
			if (hasTarget>0){ cout<<" sth wrong in line "   << __LINE__ << std::endl; }

			file_size += fread((void*)(box.tblr),   sizeof(float), 4, fp2d);
			scene->objects_2d_full.push_back(box);
			file_size += fread((void*)(&hasTarget), sizeof(uint8_t),   1, fp2d);
			if (hasTarget>0){ cout<<" sth wrong in line "  << __LINE__ << std::endl; }
      		}
		delete scene;
		fclose(myfile);
    	}
    	fclose(fp2d);
}











//Main function to test stuff
int main(){
/*	
	initTSDF(0, 10000);
	int temp=0, r=0;
	while (r>=0) r=getNextTSDF(temp);
	freeTSDF();
*/

	

	return 1;
}
