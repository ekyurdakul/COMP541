//Include
#include "tsdf_base.h"

//Global variables
std::vector<Scene3D*> scenes;
std::vector<int> box_id;
int totalObjectCount = 0;
float scale = 100;
float context_pad =3;
std::vector<int> grid_size {3,30,30,30};
int encode_type =100;
int totalScenes = 0;
string file_list = "";
string data_root =  "..//data//";
string output_data = "..//data//julia_data//";
int maxscenes = 0;
//list file
FILE* fp = NULL;

//Main function
int main(){
	cout << "Loaded my custom library." << endl;
	//New load 2d boxes
    int counter = 0;
	string file_list = data_root+"boxes2d_NYU_po_nb2000.list";

    cout<<"loading file "<<file_list<<"\n";
    FILE* fp = fopen(file_list.c_str(),"rb");
    if (fp==NULL) { std::cout<<"fail to open file: "<<file_list<<std::endl; exit(EXIT_FAILURE); }

/*
    while (feof(fp)==0) {
      Scene3D* scene = new Scene3D();
      unsigned int len = 0;
      size_t file_size = 0;
      file_size += fread((void*)(&len), sizeof(unsigned int), 1, fp);    
      if (len==0) break;
      scene->filename.resize(len);
      if (len>0) file_size += fread((void*)(scene->filename.data()), sizeof(char), len, fp);
     

      file_size += fread((void*)(scene->R), sizeof(float), 9, fp);
      file_size += fread((void*)(scene->K), sizeof(float), 9, fp);
      file_size += fread((void*)(&scene->height), sizeof(unsigned int), 1, fp);
      file_size += fread((void*)(&scene->width), sizeof(unsigned int), 1, fp); 
      file_size += fread((void*)(&len),    sizeof(unsigned int),   1, fp);
      scene->objects.resize(len);
      
      //std::cout<<scene->filename <<std::endl; 
      //std::cout<<len <<std::endl; 
      //std::cin.ignore();
      for (int bid = 0;bid<len;bid++){
	 Box2D box;
	 file_size += fread((void*)(&(box.category)), sizeof(unsigned int),   1, fp);
	 file_size += fread((void*)(box.tblr),        sizeof(float), 4, fp);
	 scene->objects_2d_tight.push_back(box);
	 
	 uint8_t hasTarget = 0;
	 file_size += fread((void*)(&hasTarget), sizeof(uint8_t),   1, fp);
	 if (hasTarget>0){
	  std::cout<<" sth wrong in line "   << __LINE__ << std::endl;
	 }

	 file_size += fread((void*)(box.tblr),   sizeof(float), 4, fp);
	 scene->objects_2d_full.push_back(box);
	 file_size += fread((void*)(&hasTarget), sizeof(uint8_t),   1, fp);
	 if (hasTarget>0){
	  std::cout<<" sth wrong in line "   << __LINE__ << std::endl;
	 }
      }
      scenes.push_back(scene);
    }
    fclose(fp);
*/
	return 1;
}

//Export functions to be called from Julia
extern "C"
{
	//Initialize variables
	void initTSDF(int db, int maxs)
	{
		if (db == 0)
			file_list = "..//data//boxes_NYU_po_test_nb2000_fb.list";
		else
			file_list = "..//data//boxes_SUNrgbd_po_test_nb2000_fb.list";

		cout << "Loading file: " << file_list << endl << endl;
		fp = fopen(file_list.c_str(),"rb");
		if (fp==NULL) { cout << "Failed to open file: "<< file_list << endl; exit(EXIT_FAILURE); }
		maxscenes = maxs;
	}

	//Call this when done with TSDF
	void freeTSDF()
	{
		fclose(fp);
	}

	//Compute TSDF for the next scene
	int getNextTSDF(int& count)
	{
		if (feof(fp)==0 && totalScenes < maxscenes)
		{
			Scene3D* scene = new Scene3D();
			unsigned int len = 0;
			fread((void*)(&len), sizeof(unsigned int), 1, fp);    
			if (len==0) return -1;
			scene->filename.resize(len);
			if (len>0) fread((void*)(scene->filename.data()), sizeof(char), len, fp);

			//TSDF file
			string tsdffile = output_data+"temp.tdsf";
			scene->filename = data_root+scene->filename+".bin";

			fread((void*)(scene->R), sizeof(float), 9, fp);
			fread((void*)(scene->K), sizeof(float), 9, fp);
			fread((void*)(&scene->height), sizeof(unsigned int), 1, fp);  
			fread((void*)(&scene->width), sizeof(unsigned int), 1, fp); 


			fread((void*)(&len),    sizeof(unsigned int),   1, fp);
			scene->objects.resize(len);
			if (len>0){
			  totalObjectCount += len;
			  for (int i=0;i<len;++i){
			      Box3D box;
			      fread((void*)(&(box.category)), sizeof(unsigned int),   1, fp);
			      fread((void*)(box.base),        sizeof(float), 9, fp);
			      fread((void*)(box.center),      sizeof(float), 3, fp);
			      fread((void*)(box.coeff),       sizeof(float), 3, fp);
			      //process box pad contex oreintation 
			      box = processbox (box, context_pad, grid_size[1]);
			      scene->objects[i]=box;
			      box_id.push_back(i);
			  }
			}
			scenes.push_back(scene);
			totalScenes++;

			cout << "Scene: " << totalScenes << " Boxes: " << len << " Bin: " << scene->filename << " TSDF: " << tsdffile << endl << endl;

			unsigned long long  time0,time1;

			time0 = get_timestamp_dss();
			float* dataCPUmem = new float[len*3*30*30*30];
			StorageT* dataGPUmem;
			checkCUDA(__LINE__, hipMalloc(&dataGPUmem, (len)*3*30*30*30*sizeof(float)));
			time1 = get_timestamp_dss();
			cout << "cpu->gpu time " << (time1-time0)/1000 << " ms" << endl;

			time0 = get_timestamp_dss();
			compute_TSDF(&scenes, &box_id, dataGPUmem,grid_size,encode_type,scale);
			time1 = get_timestamp_dss();
			cout << "compute time " << (time1-time0)/1000 << " ms" << endl;

			time0 = get_timestamp_dss();
			checkCUDA(__LINE__, hipMemcpy(dataCPUmem, dataGPUmem,(len)*3*30*30*30*sizeof(float), hipMemcpyDeviceToHost) );
			time1 = get_timestamp_dss();
			cout << "gpu->cpu time " << (time1-time0)/1000 << " ms" << endl;

			//clear for workaround
			scenes.clear();
			box_id.clear();

			//write TSDF to temp file because couldnt figure out how to pass it to Julia
			time0 = get_timestamp_dss();
			FILE * fid = fopen(tsdffile.c_str(),"wb");
			fwrite(dataCPUmem,sizeof(float),len*3*30*30*30,fid);
			fclose(fid);
			time1 = get_timestamp_dss();
			cout << "cpu->file " << (time1-time0)/1000 << " ms" << endl << endl;

			//Pass by reference paramater update
			count = len;
	
			//free memory
			delete scene;
			delete[] dataCPUmem;
			hipFree(dataGPUmem);
			return 1;
		}
		return -1;
	}
}
