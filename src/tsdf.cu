//Include
#include "tsdf.h"

//Global variables
std::vector<Scene3D*> scenes;
std::vector<int> box_id;
int totalObjectCount = 0;
float scale = 100;
float context_pad =3;
std::vector<int> grid_size {3,30,30,30};
int encode_type =100;
int totalScenes = 0;
string file_list = "..//data//boxes_NYU_po_test_nb2000_fb.list";
string data_root =  "..//data//";
string output_data = "..//data//julia_data//";

int main(int argc, char **argv){

	int requestedScene = atoi(argv[1]);
	

	FILE* fp = NULL;
	cout << "Loading file: " << file_list << endl << endl;
	fp = fopen(file_list.c_str(),"rb");
	if (fp==NULL) { cout << "Failed to open file: "<< file_list << endl; exit(EXIT_FAILURE); }


	while (feof(fp)==0)
	{
		Scene3D* scene = new Scene3D();
		unsigned int len = 0;
		fread((void*)(&len), sizeof(unsigned int), 1, fp);    
		if (len==0) return -1;
		scene->filename.resize(len);
		if (len>0) fread((void*)(scene->filename.data()), sizeof(char), len, fp);

		
		string s = scene->filename;
		scene->filename = data_root+scene->filename+".bin";

		fread((void*)(scene->R), sizeof(float), 9, fp);
		fread((void*)(scene->K), sizeof(float), 9, fp);
		fread((void*)(&scene->height), sizeof(unsigned int), 1, fp);  
		fread((void*)(&scene->width), sizeof(unsigned int), 1, fp); 


		fread((void*)(&len),    sizeof(unsigned int),   1, fp);
		scene->objects.resize(len);
		if (len>0){
		  totalObjectCount += len;
		  for (int i=0; i<len; ++i){
		      Box3D box;
		      fread((void*)(&(box.category)), sizeof(unsigned int),   1, fp);
		      fread((void*)(box.base),        sizeof(float), 9, fp);
		      fread((void*)(box.center),      sizeof(float), 3, fp);
		      fread((void*)(box.coeff),       sizeof(float), 3, fp);
		      //process box pad contex oreintation 
		      box = processbox (box, context_pad, grid_size[1]);
		      scene->objects[i]=box;
		      box_id.push_back(i);
		  }
		}
		scenes.push_back(scene);
		totalScenes++;

		if (totalScenes != requestedScene)
		{
			scenes.clear();
			box_id.clear();
			delete scene;
			continue;
		}

		cout << "Scene: " << totalScenes << " Boxes: " << len << " Bin: " << scene->filename << endl << endl;

		//Output files
		FILE* tempname = fopen("..//data//julia_data//temp.txt", "w");
		fprintf(tempname, "%s", s.substr(20).c_str());
		fclose(tempname);
		string tsdffile = output_data+"temp.tdsf";

		unsigned long long  time0,time1;

		time0 = get_timestamp_dss();
		float* dataCPUmem = new float[len*3*30*30*30];
		StorageT* dataGPUmem;
		checkCUDA(__LINE__, hipMalloc(&dataGPUmem, (len)*3*30*30*30*sizeof(float)));
		time1 = get_timestamp_dss();
		cout << "cpu->gpu time " << (time1-time0)/1000 << " ms" << endl;

		time0 = get_timestamp_dss();
		compute_TSDF(&scenes, &box_id, dataGPUmem,grid_size,encode_type,scale);
		time1 = get_timestamp_dss();
		cout << "compute time " << (time1-time0)/1000 << " ms" << endl;

		time0 = get_timestamp_dss();
		checkCUDA(__LINE__, hipMemcpy(dataCPUmem, dataGPUmem,(len)*3*30*30*30*sizeof(float), hipMemcpyDeviceToHost) );
		time1 = get_timestamp_dss();
		cout << "gpu->cpu time " << (time1-time0)/1000 << " ms" << endl;

		//write TSDF to temp file because couldnt figure out how to pass it to Julia
		time0 = get_timestamp_dss();
		FILE * fid = fopen(tsdffile.c_str(),"wb");
		fwrite(dataCPUmem,sizeof(float),len*3*30*30*30,fid);
		fclose(fid);
		time1 = get_timestamp_dss();
		cout << "cpu->file " << (time1-time0)/1000 << " ms" << endl << endl;

		//clear for workaround
		scenes.clear();
		box_id.clear();

		//free memory
		delete scene;
		delete[] dataCPUmem;
		hipFree(dataGPUmem);
		
		//Dont calculate others
		break;
	}

	
	fclose(fp);
	return 0;
}


/* USED THIS CODE TO EXTRACT DATA ALREADY
void convertBoxesList()
{
	string box2d = data_root+"boxes2d_NYU_po_nb2000.list";
    	cout<<"Loading 2d boxes list file: "<< box2d << endl;
    	FILE* fp2d = fopen(box2d.c_str(),"rb");
    	if (fp2d==NULL) { cout << "Failed to open file: "<< box2d<< endl; exit(EXIT_FAILURE); }


    	while (feof(fp2d)==0) {
      		Scene3D* scene = new Scene3D();
      		unsigned int len = 0;
      		size_t file_size = 0;
      		file_size += fread((void*)(&len), sizeof(unsigned int), 1, fp2d);    
      		if (len==0) break;
      		scene->filename.resize(len);
      		if (len>0) file_size += fread((void*)(scene->filename.data()), sizeof(char), len, fp2d);
     
		int inx = scene->filename.find_last_of("/");
		string output="Boxes//"+scene->filename.substr(inx+1)+".txt";
		FILE* myfile = fopen(output.c_str(), "w");
	

	      	file_size += fread((void*)(scene->R), sizeof(float), 9, fp2d);
		file_size += fread((void*)(scene->K), sizeof(float), 9, fp2d);
		file_size += fread((void*)(&scene->height), sizeof(unsigned int), 1, fp2d);
		file_size += fread((void*)(&scene->width), sizeof(unsigned int), 1, fp2d); 
		file_size += fread((void*)(&len),    sizeof(unsigned int),   1, fp2d);
		scene->objects.resize(len);
      
      		//cout<<scene->filename << endl;
      		for (int bid = 0;bid<len;bid++){
			//struct Box2D{
			  //unsigned int category;
			  //float tblr[4];
			//};
			Box2D box;
			file_size += fread((void*)(&(box.category)), sizeof(unsigned int),   1, fp2d);
			file_size += fread((void*)(box.tblr),        sizeof(float), 4, fp2d);
			scene->objects_2d_tight.push_back(box);
			 

			fprintf(myfile, "%d %f %f %f %f\n", box.category, box.tblr[0], box.tblr[1], box.tblr[2], box.tblr[3]);


		 	uint8_t hasTarget = 0;
			file_size += fread((void*)(&hasTarget), sizeof(uint8_t),   1, fp2d);
			if (hasTarget>0){ cout<<" sth wrong in line "   << __LINE__ << std::endl; }

			file_size += fread((void*)(box.tblr),   sizeof(float), 4, fp2d);
			scene->objects_2d_full.push_back(box);
			file_size += fread((void*)(&hasTarget), sizeof(uint8_t),   1, fp2d);
			if (hasTarget>0){ cout<<" sth wrong in line "  << __LINE__ << std::endl; }
      		}
		delete scene;
		fclose(myfile);
    	}
    	fclose(fp2d);
}
*/
