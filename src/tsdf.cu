//Compilation:
//nvcc -std=c++11 -o tsdf tsdf.cu

#include "tsdf_base.h"

/*
{
	"type": "Scene3DData",
	"name": "dataTest",
	"phase": "Testing",
	"data_root": "/home/shurans/deepDetectLocal/sunrgbd_dss_data/",
	"file_list": "/n/fs/modelnet/deepDetect/Release/result/proposal//RPN_NYU//boxes_NYU_po_test_nb2000_fb.list",
	"grid_size": [3,30,30,30],
	"batch_size": [288,96],
	"num_categories": 20,
	"bb_param_weight": [1,1,1,1,1,1],
	"encode_type": 100,
	"scale": 100,
	"box_reg": true,
	"context_pad": 3,
	"out": [
		"data",
		"label",
		"bb_tar_diff",
		"bb_loss_weights"
	],
	"GPU": 0,
	"num_percate": 0,
	"is_render": false,
	"is_combineimg": false,
	"is_combinehha": false,
	"img_fea_folder": "/home/shurans/deepDetectLocal/image_fea/RPN_NYU/po/",
	"imgfea_dim": 4096,
	"box_2dreg": false,
	"orein_cls": false,
}
*/

void compute_TSDF(string binfile, Box3D box, StorageT* datamem, vector<int> grid_size, int encode_type, float scale) {
	//compute_TSDF(box, dataGPUmem, grid_size,encode_type,scale);
    float tsdf_size = grid_size[1];
    if (grid_size[1]!=grid_size[2]||grid_size[1]!=grid_size[3]){
        cerr << "grid_size[1]!=grid_size[2]||grid_size[1]!=grid_size[3]" << endl;
        exit(EXIT_FAILURE);
    }
    int THREADS_NUM = 1024;
    int BLOCK_NUM = int((tsdf_size*tsdf_size*tsdf_size + size_t(THREADS_NUM) - 1) / THREADS_NUM);
    float* bb3d_data;
    hipMalloc(&bb3d_data,  sizeof(float)*15);
    

//Scene


    int filesize =0;
	string filename=binfile;
      std::cout<< "loading image "<< filename<<std::endl;
/*

      FILE* fp = fopen(filename.c_str(),"rb");
      if (fp==NULL) { std::cout<<"in load() :fail to open file: "<<filename<<std::endl; exit(EXIT_FAILURE); }
      grid_range = new unsigned int[3];
      filesize += fread((void*)(grid_range), sizeof(unsigned int), 3, fp);
      
      begin_range = new float[3];
      filesize += fread((void*)(begin_range), sizeof(float), 3, fp);
      filesize += fread((void*)(&grid_delta), sizeof(float), 1, fp);

      RGBDimage = new RGBDpixel[width*height];
      filesize += fread((void*)(RGBDimage), sizeof(RGBDpixel), width*height, fp);

      filesize +=  fread((void*)(&len_beIndex), sizeof(unsigned int), 1, fp);
      beIndex   = new unsigned int [len_beIndex];
      filesize += fread((void*)(beIndex), sizeof(unsigned int), len_beIndex, fp);

      filesize +=  fread((void*)(&len_beLinIdx), sizeof(unsigned int), 1, fp);
      beLinIdx  = new unsigned int [len_beLinIdx];
      filesize += fread((void*)(beLinIdx), sizeof(unsigned int), len_beLinIdx, fp);

      filesize += fread((void*)(&len_pcIndex), sizeof(unsigned int), 1, fp);
      pcIndex   = new unsigned int [len_pcIndex];
      filesize += fread((void*)(pcIndex), sizeof(unsigned int), len_pcIndex, fp);
      fclose(fp);

      GPUdata = false;
/*
    if (!GPUdata){
       if (beIndex!=NULL){
           unsigned int* beIndexCPU = beIndex;
           //checkCUDA(__LINE__,hipDeviceSynchronize());
           checkCUDA(__LINE__, hipMalloc(&beIndex, sizeof(unsigned int)*len_beIndex));
           //checkCUDA(__LINE__,hipDeviceSynchronize());
           checkCUDA(__LINE__, hipMemcpy(beIndex, beIndexCPU,sizeof(unsigned int)*len_beIndex, hipMemcpyHostToDevice));
           delete [] beIndexCPU;
       }
       else{
           std::cout << "beIndex is NULL"<<std::endl;
       }

       if (beLinIdx!=NULL){
           unsigned int* beLinIdxCPU = beLinIdx;
           //checkCUDA(__LINE__,hipDeviceSynchronize());
           checkCUDA(__LINE__, hipMalloc(&beLinIdx, sizeof(unsigned int)*len_beLinIdx));
           //checkCUDA(__LINE__,hipDeviceSynchronize());
           checkCUDA(__LINE__, hipMemcpy(beLinIdx, beLinIdxCPU,sizeof(unsigned int)*len_beLinIdx, hipMemcpyHostToDevice));
           delete [] beLinIdxCPU;
       }
       else{
           std::cout << "beLinIdx is NULL"<<std::endl;
       }

       // make it to full matrix to skip searching 
       unsigned int * beIndexFull;
       unsigned int sz = 2*sizeof(unsigned int)*(grid_range[0]+1)*(grid_range[1]+1)*(grid_range[2]+1);
       checkCUDA(__LINE__, hipMalloc(&beIndexFull, sz));
       checkCUDA(__LINE__, hipMemset(beIndexFull, 0, sz));
       int THREADS_NUM = 1024;
       int BLOCK_NUM = int((len_beLinIdx + size_t(THREADS_NUM) - 1) / THREADS_NUM);
       fillInBeIndexFull<<<BLOCK_NUM,THREADS_NUM>>>(beIndexFull,beIndex,beLinIdx,len_beLinIdx);
       checkCUDA(__LINE__,hipGetLastError());
       checkCUDA(__LINE__, hipFree(beIndex));      beIndex = NULL;
       checkCUDA(__LINE__, hipFree(beLinIdx));     beLinIdx = NULL;
       beIndex = beIndexFull;

       if (pcIndex!=NULL){
          unsigned int* pcIndexCPU = pcIndex;
          checkCUDA(__LINE__, hipMalloc(&pcIndex, sizeof(unsigned int)*len_pcIndex));
          checkCUDA(__LINE__, hipMemcpy(pcIndex, pcIndexCPU,sizeof(unsigned int)*len_pcIndex, hipMemcpyHostToDevice));
          delete [] pcIndexCPU;
       }
       else{
           std::cout << "pcIndexCPU is NULL"<<std::endl;
       }
       

       if (RGBDimage!=NULL){
         RGBDpixel* RGBDimageCPU = RGBDimage;
         checkCUDA(__LINE__, hipMalloc(&RGBDimage, sizeof(RGBDpixel)*width*height));
         checkCUDA(__LINE__, hipMemcpy( RGBDimage, RGBDimageCPU, sizeof(RGBDpixel)*width*height, hipMemcpyHostToDevice));
         delete [] RGBDimageCPU;
       }
       else{
           std::cout << "RGBDimage is NULL"<<std::endl;
       }

       if (grid_range!=NULL){ 
          unsigned int * grid_rangeCPU = grid_range;
          checkCUDA(__LINE__, hipMalloc(&grid_range, sizeof(unsigned int)*3));
          checkCUDA(__LINE__, hipMemcpy(grid_range, grid_rangeCPU, 3*sizeof(unsigned int), hipMemcpyHostToDevice));
          delete [] grid_rangeCPU;
       }
       else{
          std::cout << "grid_range is NULL"<<std::endl;
       }

       if (begin_range!=NULL){ 
          float * begin_rangeCPU = begin_range;
          checkCUDA(__LINE__, hipMalloc(&begin_range, sizeof(float)*3));
          checkCUDA(__LINE__, hipMemcpy(begin_range, begin_rangeCPU, sizeof(float)*3, hipMemcpyHostToDevice));
          delete [] begin_rangeCPU;
       }
       else{
          std::cout << "grid_range is NULL"<<std::endl;
       }


       checkCUDA(__LINE__, hipMalloc(&K_GPU, sizeof(float)*9));
       checkCUDA(__LINE__, hipMemcpy(K_GPU, (float*)K, sizeof(float)*9, hipMemcpyHostToDevice));

      
       checkCUDA(__LINE__, hipMalloc(&R_GPU, sizeof(float)*9));
       checkCUDA(__LINE__, hipMemcpy(R_GPU, (float*)R, sizeof(float)*9, hipMemcpyHostToDevice)); 

       GPUdata = true;

    }



if (!GPUdata){
         std::cout<< "Data is not at GPU cannot compute_xyz at GPU"<<std::endl;
         FatalError(__LINE__);
      }
      if (XYZimage!=NULL){
         std::cout<< "XYZimage!=NULL"<<std::endl;
         FatalError(__LINE__);
      }
      checkCUDA(__LINE__, hipMalloc(&XYZimage, sizeof(float)*width*height*3));
      compute_xyzkernel<<<width,height>>>(XYZimage,RGBDimage,K_GPU,R_GPU);
*/
	hipMemcpy(bb3d_data, box.base, sizeof(float)*15, hipMemcpyHostToDevice);

/*
	unsigned int * grid_range = scene->grid_range;
	float* R_data = scene->R_GPU;
	float* K_data = scene->K_GPU;
	float* range  = scene->begin_range;

	RGBDpixel* RGBDimage = scene->RGBDimage;
	unsigned int* star_end_indx_data = scene->beIndex;
	unsigned int* pc_lin_indx_data = scene->pcIndex;
	float* XYZimage  = scene->XYZimage;

	 if (encode_type > 99){
	    compute_TSDFGPUbox<<<BLOCK_NUM,THREADS_NUM>>>(datamem, R_data, K_data, range, scene->grid_delta, grid_range, RGBDimage, 
		           star_end_indx_data, pc_lin_indx_data, XYZimage, bb3d_data, grid_size[1],grid_size[2],grid_size[3], grid_size[0], 
		           scene->width, scene->height, encode_type, scale);

	}
	else{
	  //compute_TSDFGPUbox_proj<<<BLOCK_NUM,THREADS_NUM>>>(datamem, R_data, K_data, RGBDimage, XYZimage, bb3d_data, grid_size[1],grid_size[2],grid_size[3], grid_size[0], scene->width, scene->height, encode_type, scale);
	}
*/
	hipDeviceSynchronize();
	hipGetLastError();
    hipFree(bb3d_data);
}



int main(){
	//boxes_NYU_po_test_nb2000_fb.list
	string file_list = "boxes_SUNrgbd_po_test_nb2000_fb.list";
    	string data_root =  "";

    	float scale =100;
	float context_pad =3;
	vector<int> grid_size {3,30,30,30};
    	int encode_type =100;

    	cout <<"loading file "<< file_list << endl;
    	FILE* fp = fopen(file_list.c_str(),"rb");
    	if (fp==NULL) { cout<< "failed to open file: "<< file_list << endl; exit(EXIT_FAILURE); }


	unsigned int totalScenes = 0;
	unsigned int totalBoxes = 0;
    while (feof(fp)==0) {
	totalScenes++;
	
      unsigned int len = 0;
      fread((void*)(&len), sizeof(unsigned int), 1, fp);    
      if (len==0) break;
	string filename = "";
      	filename.resize(len);
      if (len>0) fread((void*)(filename.data()), sizeof(char), len, fp);


	string binfile = data_root+filename+".bin";
	string tsdffile = data_root+filename+".tsdf";
	float R[9];
	float K[9];
	float height;
	float width;
      	fread((void*)(R), sizeof(float), 9, fp);
 	fread((void*)(K), sizeof(float), 9, fp);
      	fread((void*)(&height), sizeof(unsigned int), 1, fp);  
      fread((void*)(&width), sizeof(unsigned int), 1, fp); 
      

      fread((void*)(&len),    sizeof(unsigned int),   1, fp);
      if (len>0){
	FILE * fid = fopen(tsdffile.c_str(),"wb");
	cout << binfile << " " << totalScenes << " ";
          for (int i=0;i<len;++i){
		totalBoxes++;

              Box3D box;
              fread((void*)(&(box.category)), sizeof(unsigned int),   1, fp);
              fread((void*)(box.base),        sizeof(float), 9, fp);
              fread((void*)(box.center),      sizeof(float), 3, fp);
              fread((void*)(box.coeff),       sizeof(float), 3, fp);
              box = processbox (box, context_pad, grid_size[1]);

		//Compute TSDF for each box
		StorageT* dataGPUmem;
		float* dataCPUmem = new float[3*30*30*30];
		hipMalloc(&dataGPUmem, 3*30*30*30*sizeof(float));

		compute_TSDF(binfile, box, dataGPUmem, grid_size,encode_type,scale);

		hipMemcpy(dataCPUmem, dataGPUmem,3*30*30*30*sizeof(float), hipMemcpyDeviceToHost);
	    	fwrite(dataCPUmem,sizeof(float),3*30*30*30,fid);
		hipFree(dataGPUmem);
		delete[] dataCPUmem;
             
          }
	cout << totalBoxes << endl;
    	fclose(fid);
      }
    }
    fclose(fp);
	return 1;
}
