#include "hip/hip_runtime.h"
#include "tsdf_base.h"

/*
{
	"type": "Scene3DData",
	"name": "dataTest",
	"phase": "Testing",
	"data_root": "/home/shurans/deepDetectLocal/sunrgbd_dss_data/",
	"file_list": "/n/fs/modelnet/deepDetect/Release/result/proposal//RPN_NYU//boxes_NYU_po_test_nb2000_fb.list",
	"grid_size": [3,30,30,30],
	"batch_size": [288,96],
	"num_categories": 20,
	"bb_param_weight": [1,1,1,1,1,1],
	"encode_type": 100,
	"scale": 100,
	"box_reg": true,
	"context_pad": 3,
	"out": [
		"data",
		"label",
		"bb_tar_diff",
		"bb_loss_weights"
	],
	"GPU": 0,
	"num_percate": 0,
	"is_render": false,
	"is_combineimg": false,
	"is_combinehha": false,
	"img_fea_folder": "/home/shurans/deepDetectLocal/image_fea/RPN_NYU/po/",
	"imgfea_dim": 4096,
	"box_2dreg": false,
	"orein_cls": false,
}
*/


void compute_TSDF(std::vector<Scene3D*> *chosen_scenes_ptr, std::vector<int> *chosen_box_id, StorageT* datamem, std::vector<int> grid_size, int encode_type, float scale) {
/*
    // for each scene 
    int totalcounter = 0;
    float tsdf_size = grid_size[1];
    if (grid_size[1]!=grid_size[2]||grid_size[1]!=grid_size[3]){
        std::cerr << "grid_size[1]!=grid_size[2]||grid_size[1]!=grid_size[3]" <<std::endl;
        exit(EXIT_FAILURE);
    }

    int numeltsdf = grid_size[0]*tsdf_size*tsdf_size*tsdf_size;
    int THREADS_NUM = 1024;
    int BLOCK_NUM = int((tsdf_size*tsdf_size*tsdf_size + size_t(THREADS_NUM) - 1) / THREADS_NUM);
    float* bb3d_data;

    //int tmpD; hipGetDevice(&tmpD); std::cout<<"GPU at LINE "<<__LINE__<<" = "<<tmpD<<std::endl;
    //checkCUDA(__LINE__,hipDeviceSynchronize());
    checkCUDA(__LINE__, hipMalloc(&bb3d_data,  sizeof(float)*15));
    
    //unsigned long long transformtime =0;
    //unsigned long long loadtime =0;
    //unsigned long long copygputime =0;
    //unsigned int sz = 0;
    Scene3D* scene_prev = NULL;
    for (int sceneId = 0;sceneId<(*chosen_scenes_ptr).size();sceneId++){
        // caculate in CPU mode
        //compute_TSDFCPUbox(tsdf_data,&((*chosen_scenes_ptr)[sceneId]),boxId,grid_size,encode_type,scale);
        // caculate in GPU mode
        
        //unsigned long long  time0,time1,time2,time3,time4;
        Scene3D* scene = (*chosen_scenes_ptr)[sceneId];
        //int tmpD; hipGetDevice(&tmpD); std::cout<<"GPU at LINE "<<__LINE__<<" = "<<tmpD<<std::endl;
        // perpare scene
        if (scene!=scene_prev){
            if (scene_prev!=NULL){
               scene_prev -> free();
            }
            scene->loadData2XYZimage(); 
        }
        
        int boxId = (*chosen_box_id)[sceneId];
        checkCUDA(__LINE__, hipMemcpy(bb3d_data, scene->objects[boxId].base, sizeof(float)*15, hipMemcpyHostToDevice));

        unsigned int * grid_range = scene->grid_range;
        float* R_data = scene->R_GPU;
        float* K_data = scene->K_GPU;
        float* range  = scene->begin_range;
        
        RGBDpixel* RGBDimage = scene->RGBDimage;
        unsigned int* star_end_indx_data = scene->beIndex;
        unsigned int* pc_lin_indx_data = scene->pcIndex;
        float* XYZimage  = scene->XYZimage;
        
        // output
        StorageT * tsdf_data = &datamem[totalcounter*numeltsdf];

        //time3 = get_timestamp_dss();   
        //checkCUDA(__LINE__,hipDeviceSynchronize());
         if (encode_type > 99){
            compute_TSDFGPUbox<<<BLOCK_NUM,THREADS_NUM>>>(tsdf_data, R_data, K_data, range, scene->grid_delta, grid_range, RGBDimage, 
                           star_end_indx_data, pc_lin_indx_data, XYZimage, bb3d_data, grid_size[1],grid_size[2],grid_size[3], grid_size[0], 
                           scene->width, scene->height, encode_type, scale);

        }
        else{
          //std::cout<<"compute_TSDFGPUbox_proj"<<std::endl;
          compute_TSDFGPUbox_proj<<<BLOCK_NUM,THREADS_NUM>>>(tsdf_data, R_data, K_data, RGBDimage, XYZimage,
                                                             bb3d_data, grid_size[1],grid_size[2],grid_size[3], grid_size[0], 
                                                             scene->width, scene->height, encode_type, scale);
        }
        
        checkCUDA(__LINE__,hipDeviceSynchronize());
        checkCUDA(__LINE__,hipGetLastError());
        //time4 = get_timestamp_dss();

        //

        ++totalcounter;

        scene_prev = scene;
        //loadtime += time1-time0;
        //copygputime += time2-time1;
        //transformtime += time4-time3;
    }
    checkCUDA(__LINE__, hipFree(bb3d_data));
    
    // free the loaded images
    for (int sceneId = 0;sceneId<(*chosen_scenes_ptr).size();sceneId++){
        (*chosen_scenes_ptr)[sceneId]->free();
    }
    
    
    //std::cout << "compute_TSDF: read disk " << loadtime/1000 << " ms, " << "copygputime " 
    //<< copygputime/1000 << "transform " << transformtime/1000 << " ms" <<std::endl;  
*/
}



int main(){
/*
std::string file_list = "DSS/boxfile/boxes_NYU_trainfea_debug.list";
    //std::string data_root = "DSS/sunrgbd_dss_data/";
    std::string data_root =  "/n/fs/modelnet/deepDetect/sunrgbd_dss_data/";
    std::vector<Scene3D*> scenes;

    //int count = 0;
    int object_count = 0;
    float scale =100;
    float context_pad =3;
    std::vector<int> grid_size {3,30,30,30};
    int encode_type =100;

    std::cout<<"loading file "<<file_list<<"\n";
    FILE* fp = fopen(file_list.c_str(),"rb");
    if (fp==NULL) { std::cout<<"fail to open file: "<<file_list<<std::endl; exit(EXIT_FAILURE); }
    while (feof(fp)==0) {
      Scene3D* scene = new Scene3D();
      unsigned int len = 0;
      fread((void*)(&len), sizeof(unsigned int), 1, fp);    
      if (len==0) break;
      scene->filename.resize(len);
      if (len>0) fread((void*)(scene->filename.data()), sizeof(char), len, fp);
      scene->filename = data_root+scene->filename+".bin"; 
      fread((void*)(scene->R), sizeof(float), 9, fp);
      fread((void*)(scene->K), sizeof(float), 9, fp);
      fread((void*)(&scene->height), sizeof(unsigned int), 1, fp);  
      fread((void*)(&scene->width), sizeof(unsigned int), 1, fp); 
      

      fread((void*)(&len),    sizeof(unsigned int),   1, fp);
      scene->objects.resize(len);
      if (len>0){
          for (int i=0;i<len;++i){
              Box3D box;
              fread((void*)(&(box.category)), sizeof(unsigned int),   1, fp);
              fread((void*)(box.base),        sizeof(float), 9, fp);
              fread((void*)(box.center),      sizeof(float), 3, fp);
              fread((void*)(box.coeff),       sizeof(float), 3, fp);
              //process box pad contex oreintation 
              box = processbox (box, context_pad, grid_size[1]);
              scene->objects[i]=box;

              object_count++;
              //num_categories = max(num_categories, box.category);
            
              //printf("category:%d\n",box.category);
              //printf("box.base:%f,%f,%f,%f,%f,%f\n",box.base[0],box.base[1],box.base[2],box.base[3],box.base[4],box.base[5]);
              //printf("box.base:%f,%f,%f,%f,%f,%f\n",box.base[0],box.base[1],box.base[2],box.base[3],box.base[4],box.base[5]);
              //printf("box.center:%f,%f,%f\n",box.center[0],box.center[1],box.center[2]);
              //printf("box.coeff:%f,%f,%f\n",box.coeff[0],box.coeff[1],box.coeff[2]);
             
          }
      }
      scenes.push_back(scene);

    }
    fclose(fp);

    std::vector<Scene3D*> chosen_scenes;
    std::vector<int> chosen_box_id;
    for (int i = 0;i<scenes.size();++i){
       for (int j =0; j < scenes[i]->objects.size();++j){
            chosen_scenes.push_back(scenes[i]);
            chosen_box_id.push_back(j);
       } 
    }

    
    std::cout<<"object_count:" <<object_count <<std::endl;
    float* dataCPUmem = new float[(object_count)*3*30*30*30];
    StorageT* dataGPUmem;
    checkCUDA(__LINE__, hipMalloc(&dataGPUmem, (object_count)*3*30*30*30*sizeof(float)));

    compute_TSDF(&chosen_scenes, &chosen_box_id, dataGPUmem,grid_size,encode_type,scale);
    checkCUDA(__LINE__, hipMemcpy(dataCPUmem, dataGPUmem,(object_count)*3*30*30*30*sizeof(float), hipMemcpyDeviceToHost) );
        

    std::string outputfile = "DSS/feature.bin";

    FILE * fid = fopen(outputfile.c_str(),"wb");
    fwrite(dataCPUmem,sizeof(float),(object_count)*3*30*30*30,fid);
    fclose(fid);
    return 1;
*/
	//string file_list = "..//data//boxes_NYU_po_test_nb2000_fb.list";
	string file_list = "..//data//boxes_SUNrgbd_po_test_nb2000_fb.list";
    	string data_root =  "..//data//";
	string output_data = "..//data//julia_data//";

    	float scale =100;
	float context_pad =3;
	vector<int> grid_size {3,30,30,30};
    	int encode_type =100;

    	cout <<"loading file "<< file_list << endl;
    	FILE* fp = fopen(file_list.c_str(),"rb");
    	if (fp==NULL) { cout<< "failed to open file: "<< file_list << endl; exit(EXIT_FAILURE); }

	unsigned int totalScenes = 0;
	unsigned int totalBoxes = 0;

	//limit to 5 scenes for testing
	int maxscenes = 5;

    	while (feof(fp)==0 && totalScenes < maxscenes) {	
	      	unsigned int len = 0;
	      	fread((void*)(&len), sizeof(unsigned int), 1, fp);    
	      	if (len==0) break;
		string filename = "";
	      	filename.resize(len);
	      	if (len>0) fread((void*)(filename.data()), sizeof(char), len, fp);

		int lastback = filename.find_last_of("/");
		string outputname = "";
		if (lastback > 0)
		{
			outputname = filename.substr(lastback+1);
		}
		else continue;
		string binfile = data_root+filename+".bin";
		string tsdffile = output_data+outputname+".tsdf";		

		float R[9];
		float K[9];
		float height;
		float width;
	      	fread((void*)(R), sizeof(float), 9, fp);
	 	fread((void*)(K), sizeof(float), 9, fp);
	      	fread((void*)(&height), sizeof(unsigned int), 1, fp);  
      		fread((void*)(&width), sizeof(unsigned int), 1, fp); 
      

      		fread((void*)(&len),    sizeof(unsigned int),   1, fp);
     		if (len>0){

			cout << totalScenes+1 << ": Boxes :" << len << " " << "TSDF\t" << tsdffile << "\t" << "Bin\t" << binfile << endl;

			FILE * fid = fopen(tsdffile.c_str(),"wb");
			for (int i=0;i<len;++i){
				totalBoxes++;

			      	Box3D box;
			      	fread((void*)(&(box.category)), sizeof(unsigned int),   1, fp);
			      	fread((void*)(box.base),        sizeof(float), 9, fp);
			      	fread((void*)(box.center),      sizeof(float), 3, fp);
			      	fread((void*)(box.coeff),       sizeof(float), 3, fp);
			      	box = processbox (box, context_pad, grid_size[1]);

				StorageT* dataGPUmem;
				float* dataCPUmem = new float[3*30*30*30];
				hipMalloc(&dataGPUmem, 3*30*30*30*sizeof(float));

				//Compute TSDF for each box and write it to file
				compute_TSDF(binfile, box, dataGPUmem, grid_size,encode_type,scale);

				hipMemcpy(dataCPUmem, dataGPUmem,3*30*30*30*sizeof(float), hipMemcpyDeviceToHost);
			    	fwrite(dataCPUmem,sizeof(float),3*30*30*30,fid);
				hipFree(dataGPUmem);
				delete[] dataCPUmem;
             
          		}//for boxes
    			fclose(fid);
      		}

		totalScenes++;
    	}//while feof scenes

	fclose(fp);
	return 1;
}
