//Include
#include "tsdf_base.h"

//Global variables
std::vector<Scene3D*> scenes;
std::vector<int> box_id;
int totalObjectCount = 0;
float scale = 100;
float context_pad =3;
std::vector<int> grid_size {3,30,30,30};
int encode_type =100;
int totalScenes = 0;
string file_list = "";
string data_root =  "..//data//";
string output_data = "..//data//julia_data//";
int maxscenes = 0;
//list file
FILE* fp = NULL;

//Main function
int main(){
	cout << "Loaded my custom library." << endl;
	return 1;
}

//Export functions to be called from Julia
extern "C"
{
	//Initialize variables
	void initTSDF(int db, int maxs)
	{
		if (db == 0)
			file_list = "..//data//boxes_NYU_po_test_nb2000_fb.list";
		else
			file_list = "..//data//boxes_SUNrgbd_po_test_nb2000_fb.list";

		cout << "Loading file: " << file_list << endl << endl;
		fp = fopen(file_list.c_str(),"rb");
		if (fp==NULL) { cout << "Failed to open file: "<< file_list << endl; exit(EXIT_FAILURE); }
		maxscenes = maxs;
	}

	//Call this when done with TSDF
	void freeTSDF()
	{
		fclose(fp);
	}

	//Compute TSDF for the next scene
	int getNextTSDF(int& count)
	{
		if (feof(fp)==0 && totalScenes < maxscenes)
		{
			Scene3D* scene = new Scene3D();
			unsigned int len = 0;
			fread((void*)(&len), sizeof(unsigned int), 1, fp);    
			if (len==0) return -1;
			scene->filename.resize(len);
			if (len>0) fread((void*)(scene->filename.data()), sizeof(char), len, fp);

			//TSDF file
			string tsdffile = output_data+"temp.tdsf";
			scene->filename = data_root+scene->filename+".bin";

			fread((void*)(scene->R), sizeof(float), 9, fp);
			fread((void*)(scene->K), sizeof(float), 9, fp);
			fread((void*)(&scene->height), sizeof(unsigned int), 1, fp);  
			fread((void*)(&scene->width), sizeof(unsigned int), 1, fp); 


			fread((void*)(&len),    sizeof(unsigned int),   1, fp);
			scene->objects.resize(len);
			if (len>0){
			  totalObjectCount += len;
			  for (int i=0;i<len;++i){
			      Box3D box;
			      fread((void*)(&(box.category)), sizeof(unsigned int),   1, fp);
			      fread((void*)(box.base),        sizeof(float), 9, fp);
			      fread((void*)(box.center),      sizeof(float), 3, fp);
			      fread((void*)(box.coeff),       sizeof(float), 3, fp);
			      //process box pad contex oreintation 
			      box = processbox (box, context_pad, grid_size[1]);
			      scene->objects[i]=box;
			      box_id.push_back(i);
			  }
			}
			scenes.push_back(scene);
			totalScenes++;

			cout << "Scene: " << totalScenes << " Boxes: " << len << " Bin: " << scene->filename << " TSDF: " << tsdffile << endl << endl;

			unsigned long long  time0,time1;

			time0 = get_timestamp_dss();
			float* dataCPUmem = new float[len*3*30*30*30];
			StorageT* dataGPUmem;
			checkCUDA(__LINE__, hipMalloc(&dataGPUmem, (len)*3*30*30*30*sizeof(float)));
			time1 = get_timestamp_dss();
			cout << "cpu->gpu time " << (time1-time0)/1000 << " ms" << endl;

			time0 = get_timestamp_dss();
			compute_TSDF(&scenes, &box_id, dataGPUmem,grid_size,encode_type,scale);
			time1 = get_timestamp_dss();
			cout << "compute time " << (time1-time0)/1000 << " ms" << endl;

			time0 = get_timestamp_dss();
			checkCUDA(__LINE__, hipMemcpy(dataCPUmem, dataGPUmem,(len)*3*30*30*30*sizeof(float), hipMemcpyDeviceToHost) );
			time1 = get_timestamp_dss();
			cout << "gpu->cpu time " << (time1-time0)/1000 << " ms" << endl;

			//clear for workaround
			scenes.clear();
			box_id.clear();

			//write TSDF to temp file because couldnt figure out how to pass it to Julia
			time0 = get_timestamp_dss();
			FILE * fid = fopen(tsdffile.c_str(),"wb");
			fwrite(dataCPUmem,sizeof(float),len*3*30*30*30,fid);
			fclose(fid);
			time1 = get_timestamp_dss();
			cout << "cpu->file " << (time1-time0)/1000 << " ms" << endl << endl;

			//Pass by reference paramater update
			count = len;
	
			//free memory
			delete scene;
			delete[] dataCPUmem;
			hipFree(dataGPUmem);
			return 1;
		}
		return -1;
	}
}
