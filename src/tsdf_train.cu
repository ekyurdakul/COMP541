//Include
#include "tsdf.h"

//Global variables
std::vector<Scene3D*> scenes;
std::vector<int> box_id;
int totalObjectCount = 0;
float scale = 100;
float context_pad =3;
std::vector<int> grid_size {3,30,30,30};
int encode_type =100;
int totalScenes = 0;
string file_list = "..//data//boxes_NYU_po_train_diff_nb2000_fb.list";
string data_root =  "..//data//";
string output_data = "..//data//julia_data//";

int main(int argc, char **argv){
	int requestedScene = atoi(argv[1]);
	

	FILE* fp = NULL;
	cout << "Loading file: " << file_list << endl << endl;
	fp = fopen(file_list.c_str(),"rb");
	if (fp==NULL) { cout << "Failed to open file: "<< file_list << endl; exit(EXIT_FAILURE); }


	while (feof(fp)==0)
	{
		Scene3D* scene = new Scene3D();
		unsigned int len = 0;
		fread((void*)(&len), sizeof(unsigned int), 1, fp);    
		if (len==0) return -1;
		scene->filename.resize(len);
		if (len>0) fread((void*)(scene->filename.data()), sizeof(char), len, fp);

		
		string s = scene->filename;
		scene->filename = data_root+scene->filename+".bin";

		fread((void*)(scene->R), sizeof(float), 9, fp);
		fread((void*)(scene->K), sizeof(float), 9, fp);
		fread((void*)(&scene->height), sizeof(unsigned int), 1, fp);  
		fread((void*)(&scene->width), sizeof(unsigned int), 1, fp); 


		fread((void*)(&len),    sizeof(unsigned int),   1, fp);
		scene->objects.resize(len);
		if (len>0){
		  totalObjectCount += len;
		  for (int i=0; i<len; ++i){
		      Box3D box;
		      fread((void*)(&(box.category)), sizeof(unsigned int),   1, fp);
		      fread((void*)(box.base),        sizeof(float), 9, fp);
		      fread((void*)(box.center),      sizeof(float), 3, fp);
		      fread((void*)(box.coeff),       sizeof(float), 3, fp);
		      //process box pad contex oreintation 
		      box = processbox (box, context_pad, grid_size[1]);
		      scene->objects[i]=box;
		      box_id.push_back(i);

			    uint8_t hasTarget = 0;
			    fread((void*)(&hasTarget), sizeof(uint8_t),   1, fp);
			    if (hasTarget>0){
			      float box_tar_diff[6];
			      fread((void*)(box_tar_diff), sizeof(float), 6, fp);
			    }
		  }
		}
		scenes.push_back(scene);
		totalScenes++;

		if (totalScenes != requestedScene)
		{
			scenes.clear();
			box_id.clear();
			delete scene;
			continue;
		}

		cout << "Scene: " << totalScenes << " Boxes: " << len << " Bin: " << scene->filename << endl << endl;

		//Output files
		FILE* tempname = fopen("..//data//julia_data//temp.txt", "w");
		fprintf(tempname, "%s", s.substr(20).c_str());
		fclose(tempname);
		string tsdffile = output_data+"temp.tdsf";

		unsigned long long  time0,time1;

		time0 = get_timestamp_dss();
		float* dataCPUmem = new float[len*3*30*30*30];
		StorageT* dataGPUmem;
		checkCUDA(__LINE__, hipMalloc(&dataGPUmem, (len)*3*30*30*30*sizeof(float)));
		time1 = get_timestamp_dss();
		cout << "cpu->gpu time " << (time1-time0)/1000 << " ms" << endl;

		time0 = get_timestamp_dss();
		compute_TSDF(&scenes, &box_id, dataGPUmem,grid_size,encode_type,scale);
		time1 = get_timestamp_dss();
		cout << "compute time " << (time1-time0)/1000 << " ms" << endl;

		time0 = get_timestamp_dss();
		checkCUDA(__LINE__, hipMemcpy(dataCPUmem, dataGPUmem,(len)*3*30*30*30*sizeof(float), hipMemcpyDeviceToHost) );
		time1 = get_timestamp_dss();
		cout << "gpu->cpu time " << (time1-time0)/1000 << " ms" << endl;

		//write TSDF to temp file because couldnt figure out how to pass it to Julia
		time0 = get_timestamp_dss();
		FILE * fid = fopen(tsdffile.c_str(),"wb");
		fwrite(dataCPUmem,sizeof(float),len*3*30*30*30,fid);
		fclose(fid);
		time1 = get_timestamp_dss();
		cout << "cpu->file " << (time1-time0)/1000 << " ms" << endl << endl;

		//clear for workaround
		scenes.clear();
		box_id.clear();

		//free memory
		delete scene;
		delete[] dataCPUmem;
		hipFree(dataGPUmem);
		
		//Dont calculate others
		break;
	}

	
	fclose(fp);
	return 0;
}
