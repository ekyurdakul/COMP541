#include <iostream>
#include <string>
#include <cstdlib>
#include <stdio.h>
#include <vector>
#include <hip/hip_runtime.h>
#include <sys/time.h>
using namespace std;

#define StorageT float
#define ComputeT float
#define CPUStorage2ComputeT(x) (x)
#define CPUCompute2StorageT(x) (x)
#define GPUStorage2ComputeT(x) (x)
#define GPUCompute2StorageT(x) (x)

enum Scene3DType { RGBD, Render, Mesh };

struct RGBDpixel{
  uint8_t R;
  uint8_t G;
  uint8_t B;
  uint8_t D;
  uint8_t D_;
};
struct Box3D{
  unsigned int category;
  float base[9];
  float center[3];
  float coeff[3];
};

struct Box2D{
  unsigned int category;
  float tblr[4];
};

unsigned long long get_timestamp_dss(){
  struct timeval now;
  gettimeofday (&now, NULL);
  return  now.tv_usec + (unsigned long long)now.tv_sec * 1000000;
};

void FatalError(const int lineNumber=0) {
	std::cerr << "FatalError";
	if (lineNumber!=0) std::cerr<<" at LINE "<<lineNumber;
	std::cerr << ". Program Terminated." << std::endl;
	hipDeviceReset();
	exit(EXIT_FAILURE);
}

void checkCUDA(const int lineNumber, hipError_t status) {
	if (status != hipSuccess) {
		std::cerr << "CUDA failure at LINE " << lineNumber << ": " << status << std::endl;
		FatalError();
	}
}

Box3D processbox(Box3D box,float context_pad,int tsdf_size){
     if (context_pad > 0){
        float context_scale = float(tsdf_size) / (float(tsdf_size) - 2*context_pad);
        box.coeff[0] = box.coeff[0] * context_scale;
        box.coeff[1] = box.coeff[1] * context_scale;
        box.coeff[2] = box.coeff[2] * context_scale;
     }
     // change the oreintation 
     if (box.base[1]<0){
        box.base[0] = -1*box.base[0];
        box.base[1] = -1*box.base[1];
        box.base[2] = -1*box.base[2];
     }
     if (box.base[4]<0){
        box.base[3] = -1*box.base[3];
        box.base[4] = -1*box.base[4];
        box.base[5] = -1*box.base[5];
     }

     if(box.base[1]<box.base[4]){
        // swap first two row 
        float tmpbase[3];
        tmpbase[0] = box.base[0];
        tmpbase[1] = box.base[1];
        tmpbase[2] = box.base[2];

        box.base[0] = box.base[3];
        box.base[1] = box.base[4];
        box.base[2] = box.base[5];

        box.base[3] = tmpbase[0];
        box.base[4] = tmpbase[1];
        box.base[5] = tmpbase[2];
        float tmpcoeff =  box.coeff[0];
        box.coeff[0] = box.coeff[1];
        box.coeff[1] = tmpcoeff;
     }
     return box;
}

void __global__ compute_xyzkernel(float * XYZimage, RGBDpixel * RGBDimage, float * K, float * R){
            int ix = blockIdx.x;
            int iy = threadIdx.x;
            int height = blockDim.x;
            //
            uint16_t D = (uint16_t)RGBDimage[iy + ix * height].D;
            uint16_t D_ = (uint16_t)RGBDimage[iy + ix * height].D_;
            D_ = D_<<8;
            float depth = float(D|D_)/1000.0;
            
            // project the depth point to 3d
            float tdx = (float(ix + 1) - K[2]) * depth / K[0];
            float tdz =  - (float(iy + 1) - K[5]) * depth / K[4];
            float tdy = depth;

            XYZimage[3 * (iy + ix * height) + 0] = R[0] * tdx + R[1] * tdy + R[2] * tdz;
            XYZimage[3 * (iy + ix * height) + 1] = R[3] * tdx + R[4] * tdy + R[5] * tdz;
            XYZimage[3 * (iy + ix * height) + 2] = R[6] * tdx + R[7] * tdy + R[8] * tdz;

}

void __global__ fillInBeIndexFull(unsigned int* beIndexFull, unsigned int* beIndex, unsigned int* beLinIdx, unsigned int len_beLinIdx){
     const int index = threadIdx.x + blockIdx.x * blockDim.x;
     if (index>=len_beLinIdx) {
        return;
     }
     else{
        beIndexFull[2*beLinIdx[index]+0] =  beIndex[2*index+0];
        beIndexFull[2*beLinIdx[index]+1] =  beIndex[2*index+1];
     }
}

__global__ void compute_TSDFGPUbox(StorageT* tsdf_data, float* R_data, float* K_data,  float* range, float grid_delta,  unsigned int *grid_range,
                                  RGBDpixel* RGBDimage,  unsigned int* star_end_indx_data ,unsigned int*  pc_lin_indx_data,float* XYZimage,
                                  const float* bb3d_data, int tsdf_size,int tsdf_size1,int tsdf_size2, int fdim, int im_w, int im_h, const int encode_type,const float scale)
{
    const int index = threadIdx.x + blockIdx.x * blockDim.x;;
    int volume_size = tsdf_size * tsdf_size1 * tsdf_size2;
    if (index > volume_size) return;
    float delta_x = 2 * bb3d_data[12] / float(tsdf_size);  
    float delta_y = 2 * bb3d_data[13] / float(tsdf_size1);  
    float delta_z = 2 * bb3d_data[14] / float(tsdf_size2);  
    float surface_thick = 0.1;
    const float MaxDis = surface_thick + 20;
    //printf("delta_x:%f,%f,%f\n",R_data[0],R_data[1],R_data[2]); 
    // caculate tsdf for this box
    /*
    float x = float(index % tsdf_size);
    float y = float((index / tsdf_size) % tsdf_size);   
    float z = float((index / tsdf_size / tsdf_size) % tsdf_size);
    */
    float x = float((index / (tsdf_size1*tsdf_size2))%tsdf_size) ;
    float y = float((index / tsdf_size2) % tsdf_size1);
    float z = float(index % tsdf_size2);

    for (int i =0;i<fdim;i++){
        tsdf_data[index + i * volume_size] = GPUCompute2StorageT(ComputeT(0));
    }

    // get grid world coordinate
    float temp_x = - bb3d_data[12] + (x + 0.5) * delta_x;
    float temp_y = - bb3d_data[13] + (y + 0.5) * delta_y;
    float temp_z = - bb3d_data[14] + (z + 0.5) * delta_z;

    x = temp_x * bb3d_data[0] + temp_y * bb3d_data[3] + temp_z * bb3d_data[6]
        + bb3d_data[9];
    y = temp_x * bb3d_data[1] + temp_y * bb3d_data[4] + temp_z * bb3d_data[7]
        + bb3d_data[10];
    z = temp_x * bb3d_data[2] + temp_y * bb3d_data[5] + temp_z * bb3d_data[8]
        + bb3d_data[11]; 

    // project to image plane decides the sign
    // rotate back and swap y, z and -y
    float xx =   R_data[0] * x + R_data[3] * y + R_data[6] * z;
    float zz =   R_data[1] * x + R_data[4] * y + R_data[7] * z;
    float yy = - R_data[2] * x - R_data[5] * y - R_data[8] * z;
    int ix = floor(xx * K_data[0] / zz + K_data[2]+0.5) - 1;
    int iy = floor(yy * K_data[4] / zz + K_data[5]+0.5) - 1;

    
    if (ix < 0 || ix >= im_w || iy < 0 || iy >= im_h || zz < 0.0001){
        return;
    } 

    // find the most nearby point 
    float disTosurfaceMin = MaxDis;
    int idx_min = 0;
    int x_grid = floor((x-range[0])/grid_delta);
    int y_grid = floor((y-range[1])/grid_delta);
    int z_grid = floor((z-range[2])/grid_delta);
    //grid_range =  [w,d,h];  linearInd =x(i)*d*h+y(i)*h+z(i);
    //if (x_grid < 0 || x_grid >= grid_range[0] || y_grid < 0 || y_grid >= grid_range[1] || z_grid < 0 || z_grid >= grid_range[2]){
    if (x_grid < 0 || x_grid > grid_range[0] || y_grid < 0 || y_grid > grid_range[1] || z_grid < 0 || z_grid > grid_range[2]){
        return;
    }
    int linearInd =x_grid*grid_range[1]*grid_range[2]+y_grid*grid_range[2]+z_grid;      
    int search_region =1;
    if (star_end_indx_data[2*linearInd+0]>0){
        search_region =0;
    }  
    int find_close_point = -1;

    while(find_close_point<0&&search_region<3){
      for (int iix = max(0,x_grid-search_region); iix < min((int)grid_range[0],x_grid+search_region+1); iix++){
        for (int iiy = max(0,y_grid-search_region); iiy < min((int)grid_range[1],y_grid+search_region+1); iiy++){
          for (int iiz = max(0,z_grid-search_region); iiz < min((int)grid_range[2],z_grid+search_region+1); iiz++){
              unsigned int iilinearInd = iix*grid_range[1]*grid_range[2] + iiy*grid_range[2] + iiz;

              for (int pid = star_end_indx_data[2*iilinearInd+0]-1; pid < star_end_indx_data[2*iilinearInd+1]-1;pid++){
                 
                 //printf("%d-%d\n",star_end_indx_data[2*iilinearInd+0],star_end_indx_data[2*iilinearInd+1]);
                 unsigned int p_idx_lin = pc_lin_indx_data[pid];
                 float xp = XYZimage[3*p_idx_lin+0];
                 float yp = XYZimage[3*p_idx_lin+1];
                 float zp = XYZimage[3*p_idx_lin+2];
                 // distance
                 float xd = abs(x - xp);
                 float yd = abs(y - yp);
                 float zd = abs(z - zp);
                 if (xd < 2.0 * delta_x||yd < 2.0 * delta_x|| zd < 2.0 * delta_x){
                    float disTosurface = sqrt(xd * xd + yd * yd + zd * zd);
                    if (disTosurface < disTosurfaceMin){
                       disTosurfaceMin = disTosurface;
                       idx_min = p_idx_lin;
                       find_close_point = 1;
                       //printf("x:%f,%f,%f,xp,%f,%f,%f,xd%f,%f,%f,%f\n",x,y,z,xp,yp,zp,xd,yd,zd,disTosurfaceMin);
                       
                    }
                }
              } // for all points in this grid
            

          }
        }
      }
      search_region ++;
    }//while 
    
    float tsdf_x = MaxDis;
    float tsdf_y = MaxDis;
    float tsdf_z = MaxDis;


    float color_b =0;
    float color_g =0;
    float color_r =0;

    float xnear = 0;
    float ynear = 0;
    float znear = 0;
    if (find_close_point>0){
        
        xnear = XYZimage[3*idx_min+0];
        ynear = XYZimage[3*idx_min+1];
        znear = XYZimage[3*idx_min+2];
        tsdf_x = abs(x - xnear);
        tsdf_y = abs(y - ynear);
        tsdf_z = abs(z - znear);

        color_b = float(RGBDimage[idx_min].B)/255.0;
        color_g = float(RGBDimage[idx_min].G)/255.0;
        color_r = float(RGBDimage[idx_min].R)/255.0;

        //printf("x:%f,tsdf_x:%f,%f,%f\n",disTosurfaceMin,tsdf_x,tsdf_y,tsdf_z);          
    }
//printf("before : %f,%f,%f\n",tsdf_x,tsdf_y,tsdf_z);

    disTosurfaceMin = min(disTosurfaceMin/surface_thick,float(1.0));
    float ratio = 1.0 - disTosurfaceMin;
    float second_ratio =0;
    if (ratio > 0.5) {
       second_ratio = 1 - ratio;
    }
    else{
       second_ratio = ratio;
    }

    if (disTosurfaceMin > 0.999){
        tsdf_x = MaxDis;
        tsdf_y = MaxDis;
        tsdf_z = MaxDis;
    }

    
    if (encode_type == 101){ 
      tsdf_x = min(tsdf_x, surface_thick);
      tsdf_y = min(tsdf_y, surface_thick);
      tsdf_z = min(tsdf_z, surface_thick);
    }
    else{
      tsdf_x = min(tsdf_x, float(2.0 * delta_x));
      tsdf_y = min(tsdf_y, float(2.0 * delta_y));
      tsdf_z = min(tsdf_z, float(2.0 * delta_z));
    }

   

    float depth_project   = XYZimage[3*(ix * im_h + iy)+1];  
    if (zz > depth_project) {
      tsdf_x = - tsdf_x;
      tsdf_y = - tsdf_y;
      tsdf_z = - tsdf_z;
      disTosurfaceMin = - disTosurfaceMin;
      second_ratio = - second_ratio;
    }

    // encode_type 
    if (encode_type == 100||encode_type == 101){
      tsdf_data[index + 0 * volume_size] = GPUCompute2StorageT(tsdf_x);
      tsdf_data[index + 1 * volume_size] = GPUCompute2StorageT(tsdf_y);
      tsdf_data[index + 2 * volume_size] = GPUCompute2StorageT(tsdf_z);
    }
    else if(encode_type == 102){
      tsdf_data[index + 0 * volume_size] = GPUCompute2StorageT(tsdf_x);
      tsdf_data[index + 1 * volume_size] = GPUCompute2StorageT(tsdf_y);
      tsdf_data[index + 2 * volume_size] = GPUCompute2StorageT(tsdf_z);
      tsdf_data[index + 3 * volume_size] = GPUCompute2StorageT(color_b/scale);
      tsdf_data[index + 4 * volume_size] = GPUCompute2StorageT(color_g/scale);
      tsdf_data[index + 5 * volume_size] = GPUCompute2StorageT(color_r/scale);
    }
    else if(encode_type == 103){
      tsdf_data[index + 0 * volume_size] = GPUCompute2StorageT(ratio);
    }

    // scale feature 
    for (int i =0;i<fdim;i++){
        tsdf_data[index + i * volume_size] = GPUCompute2StorageT(scale* GPUStorage2ComputeT(tsdf_data[index + i * volume_size]));
    }

    //}// end for each index in each box
};


__global__ void compute_TSDFGPUbox_proj(StorageT* tsdf_data, float* R_data, float* K_data, RGBDpixel* RGBDimage, float* XYZimage,
                                      const float* bb3d_data, int tsdf_size,int tsdf_size1,int tsdf_size2, int fdim, int im_w, int im_h, const int encode_type,const float scale)
{
  const int index = threadIdx.x + blockIdx.x * blockDim.x;;
    int volume_size = tsdf_size * tsdf_size1 * tsdf_size2;
    if (index > volume_size) return;
    float delta_x = 2 * bb3d_data[12] / float(tsdf_size);  
    float delta_y = 2 * bb3d_data[13] / float(tsdf_size1);  
    float delta_z = 2 * bb3d_data[14] / float(tsdf_size2);  
    float surface_thick = 0.1;
    const float MaxDis = surface_thick + 20;

    float x = float((index / (tsdf_size1*tsdf_size2))%tsdf_size) ;
    float y = float((index / tsdf_size2) % tsdf_size1);
    float z = float(index % tsdf_size2);

    for (int i =0;i<fdim;i++){
        tsdf_data[index + i * volume_size] = GPUCompute2StorageT(ComputeT(0));
    }

    // get grid world coordinate
    float temp_x = - bb3d_data[12] + (x + 0.5) * delta_x;
    float temp_y = - bb3d_data[13] + (y + 0.5) * delta_y;
    float temp_z = - bb3d_data[14] + (z + 0.5) * delta_z;

    x = temp_x * bb3d_data[0] + temp_y * bb3d_data[3] + temp_z * bb3d_data[6]
        + bb3d_data[9];
    y = temp_x * bb3d_data[1] + temp_y * bb3d_data[4] + temp_z * bb3d_data[7]
        + bb3d_data[10];
    z = temp_x * bb3d_data[2] + temp_y * bb3d_data[5] + temp_z * bb3d_data[8]
        + bb3d_data[11]; 

    // project to image plane decides the sign
    // rotate back and swap y, z and -y
    float xx =   R_data[0] * x + R_data[3] * y + R_data[6] * z;
    float zz =   R_data[1] * x + R_data[4] * y + R_data[7] * z;
    float yy = - R_data[2] * x - R_data[5] * y - R_data[8] * z;
    int ix = floor(xx * K_data[0] / zz + K_data[2]+0.5) - 1;
    int iy = floor(yy * K_data[4] / zz + K_data[5]+0.5) - 1;

    
    if (ix < 0 || ix >= im_w || iy < 0 || iy >= im_h || zz < 0.0001){
        return;
    } 
    
    float x_project   = XYZimage[3*(ix * im_h + iy)+0];
    float y_project   = XYZimage[3*(ix * im_h + iy)+1];
    float z_project   = XYZimage[3*(ix * im_h + iy)+2]; 


    float tsdf_x = abs(x - x_project);
    float tsdf_y = abs(y - y_project);
    float tsdf_z = abs(z - z_project);

    float color_b = 0;
    float color_g = 0;
    float color_r = 0;
    if (RGBDimage!=NULL){
      color_b = float(RGBDimage[(ix * im_h + iy)].B)/255.0;
      color_g = float(RGBDimage[(ix * im_h + iy)].G)/255.0;
      color_r = float(RGBDimage[(ix * im_h + iy)].R)/255.0;
    }

    float disTosurfaceMin = sqrt(tsdf_x * tsdf_x + tsdf_y * tsdf_y + tsdf_z * tsdf_z);
    disTosurfaceMin = min(disTosurfaceMin/surface_thick,float(1.0));
    float ratio = 1.0 - disTosurfaceMin;
    float second_ratio =0;
    if (ratio > 0.5) {
       second_ratio = 1 - ratio;
    }
    else{
       second_ratio = ratio;
    }
    if (disTosurfaceMin > 0.999){
        tsdf_x = MaxDis;
        tsdf_y = MaxDis;
        tsdf_z = MaxDis;
    }

    tsdf_x = min(tsdf_x, float(2.0 * delta_x));
    tsdf_y = min(tsdf_y, float(2.0 * delta_y));
    tsdf_z = min(tsdf_z, float(2.0 * delta_z));

    if (zz > y_project) {
      tsdf_x = - tsdf_x;
      tsdf_y = - tsdf_y;
      tsdf_z = - tsdf_z;
      disTosurfaceMin = - disTosurfaceMin;
      second_ratio = - second_ratio;
    }

    // encode_type 
    if (encode_type == 0){
      tsdf_data[index + 0 * volume_size] = GPUCompute2StorageT(tsdf_x);
      tsdf_data[index + 1 * volume_size] = GPUCompute2StorageT(tsdf_y);
      tsdf_data[index + 2 * volume_size] = GPUCompute2StorageT(tsdf_z);
    }
    if (encode_type == 2){
      tsdf_data[index + 0 * volume_size] = GPUCompute2StorageT(tsdf_x);
      tsdf_data[index + 1 * volume_size] = GPUCompute2StorageT(tsdf_y);
      tsdf_data[index + 2 * volume_size] = GPUCompute2StorageT(tsdf_z);
      tsdf_data[index + 3 * volume_size] = GPUCompute2StorageT(color_b/scale);
      tsdf_data[index + 4 * volume_size] = GPUCompute2StorageT(color_g/scale);
      tsdf_data[index + 5 * volume_size] = GPUCompute2StorageT(color_r/scale);
    }
    // scale feature 
    for (int i =0;i<fdim;i++){
        tsdf_data[index + i * volume_size] = GPUCompute2StorageT(scale* GPUStorage2ComputeT(tsdf_data[index + i * volume_size]));
    }
}

